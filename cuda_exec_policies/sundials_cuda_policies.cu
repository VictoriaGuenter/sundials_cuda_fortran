
  #include <sundials/sundials_cuda_policies.hpp>
  #include <hip/hip_runtime.h>
 
 extern "C" {
   typedef sundials::cuda::ExecPolicy* ExecPolicyPtr;
 
   ExecPolicyPtr ThreadDirectExecPolicy_new(const size_t blockDim, hipStream_t stream = 0) {
     return new sundials::cuda::ThreadDirectExecPolicy(blockDim, stream);
   }
 
   ExecPolicyPtr GridStrideExecPolicy_new(const size_t blockDim, const size_t gridDim, hipStream_t stream = 0) {
     return new sundials::cuda::GridStrideExecPolicy(blockDim, gridDim, stream);
   }
 
   ExecPolicyPtr BlockReduceExecPolicy_new(const size_t blockDim, const size_t gridDim = 0, hipStream_t stream = 0) {
     return new sundials::cuda::BlockReduceExecPolicy(blockDim, gridDim, stream);
   }
 
   ExecPolicyPtr BlockReduceAtomicExecPolicy_new(const size_t blockDim, const size_t gridDim = 0, hipStream_t stream = 0) {
     return new sundials::cuda::BlockReduceAtomicExecPolicy(blockDim, gridDim, stream);
   }
 }
 
 
